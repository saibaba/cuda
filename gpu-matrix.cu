// https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#introduction


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <cstdio>
#include <chrono>

typedef std::chrono::high_resolution_clock Clock;

#define ITER_ROW 1024
#define ITER_COL 1024

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


void matrix_add_cpu(int *a, int *b, int *c, int m, int n) {

    // Add the vector elements a and b to the vector c
    for (int row = 0; row < m ; row++) {
        for (int col = 0; col < n ; col++) {
            int loc = n*row+col;
            *(c+loc) = *(a+loc) + *(b+loc);
        }
    }
}

__global__ void matrix_add_gpu(int *gpu_a, int *gpu_b, int *gpu_c) {
    int blockId = gridDim.x * blockIdx.y + blockIdx.x;
    int loc = blockId * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
    *(gpu_c+loc) = (*(gpu_a+loc)) + (*(gpu_b+loc));
}

void print_matrix(int *m, int rows, int cols) {
    for (int row = 0; row < rows; ++row) {
        for (int col = 0; col < cols; ++col) {
            int loc = (cols*row)+col;
            int val = *(m+loc);
            printf("\t%d", val);
        }
        printf("\n");
    }
}

int main(int argc, char **argv) {

    int rows = ITER_ROW;
    int cols = ITER_COL;

    int *a, *b, *c;
    int *gpu_a, *gpu_b, *gpu_c;
    
    size_t mem_size = rows * cols * sizeof(int);

    a = (int *)malloc(mem_size);
    b = (int *)malloc(mem_size);
    c = (int *)malloc(mem_size);

    // We need variables accessible to the GPU,
    // so cudaMallocManaged provides these
    gpuErrchk(hipMalloc(&gpu_a, mem_size));
    gpuErrchk(hipMalloc(&gpu_b, mem_size));
    gpuErrchk(hipMalloc(&gpu_c, mem_size));

    for (int row = 0; row < rows; ++row) {
        for (int col = 0; col < cols; ++col) {
            int loc = cols*row+col;
            *(a+loc) = row+col;
            *(b+loc) = row+col;
        }
    }

    gpuErrchk(hipMemcpy(gpu_a, a, mem_size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(gpu_b, b, mem_size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(gpu_c, c, mem_size, hipMemcpyHostToDevice));

    // Call the CPU function and time it
    auto cpu_start = Clock::now();
    matrix_add_cpu(a, b, c, rows, cols);
    auto cpu_end = Clock::now();
    std::cout << "vector_add_cpu: "
    << std::chrono::duration_cast<std::chrono::nanoseconds>(cpu_end - cpu_start).count()
    << " nanoseconds.\n";
    // print_matrix(c, rows, cols);
    printf("CPU (0, 0): %d\n", *c);
    printf("CPU (3, 17): %d\n", *(c+(13*cols)+17));
    printf("CPU last: %d\n", *(c+(rows*cols)-1));


    *(c+(rows*cols)-1) = 0;
    for (int row = 0; row < rows; ++row) {
      for (int col = 0; col < cols; ++col) {
        int loc = cols*row+col;
        *(c+loc) = 0;
      }
    }

    // Call the GPU function and time it
    // The triple angle brakets is a CUDA runtime extension that allows
    // parameters of a CUDA kernel call to be passed.
    // In this example, we are passing one thread block with ITER GPU threads.

    // why, these dimensions: just for fun: overall we wanted 1024x1024 threads and we can have only 1024 threads/block
    dim3 blocksPerGrid(16, 64);
    dim3 threadsPerBlock(32, 32);

    auto gpu_start = Clock::now();
    matrix_add_gpu <<<blocksPerGrid, threadsPerBlock>>> (gpu_a, gpu_b, gpu_c);
    gpuErrchk( hipPeekAtLastError() );

    gpuErrchk(hipDeviceSynchronize());
    auto gpu_end = Clock::now();
    std::cout << "vector_add_gpu: "
    << std::chrono::duration_cast<std::chrono::nanoseconds>(gpu_end - gpu_start).count()
    << " nanoseconds.\n";

    gpuErrchk(hipMemcpy(c, gpu_c, mem_size, hipMemcpyDeviceToHost));

    printf("GPU (0, 0): %d\n", *c);
    printf("GPU (3, 17): %d\n", *(c+(13*cols)+17));
    printf("GPU last: %d\n", *(c+(rows*cols)-1));
    //print_matrix(c, rows, cols);

    // Free the GPU-function based memory allocations
    hipFree(gpu_a);
    hipFree(gpu_b);
    hipFree(gpu_c);

    // Free the CPU-function based memory allocations
    free(a);
    free(b);
    free(c);

    return 0;
}
