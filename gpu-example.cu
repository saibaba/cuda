#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <cstdio>
#include <chrono>

typedef std::chrono::high_resolution_clock Clock;

#define ITER 65535

// CPU version of the vector add function
void vector_add_cpu(int *a, int *b, int *c, int n) {
    int i;

    // Add the vector elements a and b to the vector c
    for (i = 0; i < n; ++i) {
    c[i] = a[i] + b[i];
    }
}

// GPU version of the vector add function via "__global__" prefix.
// These kind of functions are called kernels in CUDA. When called, they are executed in parallel by N different threads
// as opposed to only once in a regular c++ function
__global__ void vector_add_gpu(int *gpu_a, int *gpu_b, int *gpu_c) {
    int i = threadIdx.x;
    // No for loop needed because the CUDA runtime
    // will thread this ITER times
    gpu_c[i] = gpu_a[i] + gpu_b[i];
}

int main() {

    int *a, *b, *c;
    int *gpu_a, *gpu_b, *gpu_c;

    a = (int *)malloc(ITER * sizeof(int));
    b = (int *)malloc(ITER * sizeof(int));
    c = (int *)malloc(ITER * sizeof(int));

    // We need variables accessible to the GPU,
    // so cudaMallocManaged provides these
    hipMallocManaged(&gpu_a, ITER * sizeof(int));
    hipMallocManaged(&gpu_b, ITER * sizeof(int));
    hipMallocManaged(&gpu_c, ITER * sizeof(int));

    for (int i = 0; i < ITER; ++i) {
        a[i] = i;
        b[i] = i;
        c[i] = i;
    }

    // Call the CPU function and time it
    auto cpu_start = Clock::now();
    vector_add_cpu(a, b, c, ITER);
    auto cpu_end = Clock::now();
    std::cout << "vector_add_cpu: "
    << std::chrono::duration_cast<std::chrono::nanoseconds>(cpu_end - cpu_start).count()
    << " nanoseconds.\n";

    // Call the GPU function and time it
    // The triple angle brakets is a CUDA runtime extension that allows
    // parameters of a CUDA kernel call to be passed.
    // In this example, we are passing one thread block with ITER GPU threads.
    auto gpu_start = Clock::now();
    vector_add_gpu <<<1, ITER>>> (gpu_a, gpu_b, gpu_c);
    hipDeviceSynchronize();
    auto gpu_end = Clock::now();
    std::cout << "vector_add_gpu: "
    << std::chrono::duration_cast<std::chrono::nanoseconds>(gpu_end - gpu_start).count()
    << " nanoseconds.\n";

    // Free the GPU-function based memory allocations
    hipFree(gpu_a);
    hipFree(gpu_b);
    hipFree(gpu_c);

    // Free the CPU-function based memory allocations
    free(a);
    free(b);
    free(c);

    return 0;
}
